#include "hip/hip_runtime.h"
/*************************************************************************/
/*  compute_friction.cu                                                  */
/*************************************************************************/
/*                       This file is part of:                           */
/*                           GODOT ENGINE                                */
/*                      https://godotengine.org                          */
/*************************************************************************/
/* Copyright (c) 2007-2018 Juan Linietsky, Ariel Manzur.                 */
/* Copyright (c) 2014-2018 Godot Engine contributors (cf. AUTHORS.md)    */
/*                                                                       */
/* Permission is hereby granted, free of charge, to any person obtaining */
/* a copy of this software and associated documentation files (the       */
/* "Software"), to deal in the Software without restriction, including   */
/* without limitation the rights to use, copy, modify, merge, publish,   */
/* distribute, sublicense, and/or sell copies of the Software, and to    */
/* permit persons to whom the Software is furnished to do so, subject to */
/* the following conditions:                                             */
/*                                                                       */
/* The above copyright notice and this permission notice shall be        */
/* included in all copies or substantial portions of the Software.       */
/*                                                                       */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,       */
/* EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF    */
/* MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.*/
/* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY  */
/* CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,  */
/* TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE     */
/* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.                */
/*************************************************************************/

/**
	@author AndreaCatania
*/

/**
 * This is writtein in Cuda C
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "thirdparty/flex/include/NvFlex.h"
#include "thirdparty/flex/core/maths.h"

static const int kNumThreadsPerBlock = 256;

struct GdFlexExtComputeFrictionCallback{
	NvFlexSolver *solver;

	int primitive_body_count;
	float* d_primitive_transf;
	float* d_primitive_lvelocity;
	float* d_primitive_avelocity;
	float* d_primitive_aabbs;
	float* d_primitive_extent;


	GdFlexExtComputeFrictionCallback(NvFlexSolver* p_solver)
		: solver(p_solver),
		  primitive_body_count(0),
		  d_primitive_transf(NULL),
		  d_primitive_lvelocity(NULL),
		  d_primitive_avelocity(NULL),
		  d_primitive_aabbs(NULL),
		  d_primitive_extent(NULL)
	{}

	~GdFlexExtComputeFrictionCallback(){
		free_all();
	}

	void free_all(){
		hipFree(d_primitive_transf);
		hipFree(d_primitive_lvelocity);
		hipFree(d_primitive_avelocity);
		hipFree(d_primitive_aabbs);
		hipFree(d_primitive_extent);

		d_primitive_transf = NULL;
		d_primitive_lvelocity = NULL;
		d_primitive_avelocity = NULL;
		d_primitive_aabbs = NULL;
		d_primitive_extent = NULL;
	}
};

GdFlexExtComputeFrictionCallback *GdFlexExtCreateComputeFrictionCallback(NvFlexSolver *p_solver){
	return new GdFlexExtComputeFrictionCallback(p_solver);
}

void GdFlexExtDestroyComputeFrictionCallback(GdFlexExtComputeFrictionCallback* p_callback){
	delete p_callback;
}

__device__ bool AABB_intersect(Vec3 aabb1_min, Vec3 aabb1_max, Vec3 aabb2_min, Vec3 aabb2_max) {

	if(aabb1_min.x >= aabb2_max.x)
		return false;

	if(aabb1_max.x <= aabb2_min.x)
		return false;

	if(aabb1_min.y >= aabb2_max.y)
		return false;

	if(aabb1_max.y <= aabb2_min.y)
		return false;

	if(aabb1_min.z >= aabb2_max.z)
		return false;

	if(aabb1_max.z <= aabb2_min.z)
		return false;

	return true;
}

__global__ void ComputeFriction(float dt, Vec4* __restrict__ positions, Vec4* __restrict__ velocities, int p_primitive_body_count, Vec3* p_primitive_aabbs ){

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int sorted_particle_index = i;

	float radius = 0.1;
	Vec3 radius_vec(radius, radius, radius);

	const Vec4 pos_4 = positions[sorted_particle_index];
	const Vec3 pos = Vec3(pos_4.x, pos_4.y, pos_4.z);

	Vec3 particle_aabb_begin = pos - radius_vec;
	Vec3 particle_aabb_end = pos + radius;

	const Vec4 vel_4 = velocities[sorted_particle_index];
	Vec3 velocity = Vec3(vel_4.x, vel_4.y, vel_4.z);

	for(int p = 0; p < p_primitive_body_count; p++ ){

		if( !AABB_intersect(particle_aabb_begin,
						   particle_aabb_end,
						   p_primitive_aabbs[p*2+0],
						   p_primitive_aabbs[p*2+0] + p_primitive_aabbs[p*2+1]) )
			continue;

		velocity = Vec3(0, 0, 0);
	}

	velocities[sorted_particle_index] = Vec4(velocity.x, velocity.y, velocity.z, 0);
}

void ComputeFrictionCallback(NvFlexSolverCallbackParams p_params){

	GdFlexExtComputeFrictionCallback* callback = static_cast<GdFlexExtComputeFrictionCallback*>(p_params.userData);

	const int particle_count = p_params.numActive;
	const int kNumBlocks = (particle_count + kNumThreadsPerBlock - 1) / kNumThreadsPerBlock;

	ComputeFriction<<<kNumBlocks, kNumThreadsPerBlock>>>(p_params.dt,
													(Vec4*)p_params.particles,
													(Vec4*)p_params.velocities,
													callback->primitive_body_count,
													(Vec3*)callback->d_primitive_aabbs);
}

void GdFlexExtSetComputeFrictionCallback(GdFlexExtComputeFrictionCallback* p_callback, int p_primitive_body_count, float* p_primitive_transform, float* p_primitive_lvelocity, float* p_primitive_avelocity, float* p_primitive_aabbs, float* p_primitive_extent){

	if( p_callback->primitive_body_count != p_primitive_body_count ){

		p_callback->primitive_body_count = p_primitive_body_count;

		p_callback->free_all();

		hipMalloc(&p_callback->d_primitive_transf, sizeof(float) * p_primitive_body_count * 12); // Transform
		hipMalloc(&p_callback->d_primitive_lvelocity, sizeof(float) * p_primitive_body_count * 3); // Vector3
		hipMalloc(&p_callback->d_primitive_avelocity, sizeof(float) * p_primitive_body_count * 3); // Vector3
		hipMalloc(&p_callback->d_primitive_aabbs, sizeof(float) * p_primitive_body_count * 6); // AABB
		hipMalloc(&p_callback->d_primitive_extent, sizeof(float) * p_primitive_body_count * 3); // Vector3
	}

	if(!p_primitive_body_count)
		return;

	hipMemcpy(p_callback->d_primitive_transf, p_primitive_transform, sizeof(float) * p_primitive_body_count * 12, hipMemcpyHostToDevice);
	hipMemcpy(p_callback->d_primitive_lvelocity, p_primitive_lvelocity, sizeof(float) * p_primitive_body_count * 3, hipMemcpyHostToDevice);
	hipMemcpy(p_callback->d_primitive_avelocity, p_primitive_avelocity, sizeof(float) * p_primitive_body_count * 3, hipMemcpyHostToDevice);
	hipMemcpy(p_callback->d_primitive_aabbs, p_primitive_aabbs, sizeof(float) * p_primitive_body_count * 6, hipMemcpyHostToDevice);
	hipMemcpy(p_callback->d_primitive_extent, p_primitive_extent, sizeof(float) * p_primitive_body_count * 3, hipMemcpyHostToDevice);

	NvFlexSolverCallback solver_callback;
	solver_callback.function = ComputeFrictionCallback;
	solver_callback.userData = p_callback;

	NvFlexRegisterSolverCallback(p_callback->solver, solver_callback, eNvFlexStageSubstepEnd);
}
