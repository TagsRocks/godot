#include "hip/hip_runtime.h"
/*************************************************************************/
/*  collision_verifier.cu                                                */
/*************************************************************************/
/*                       This file is part of:                           */
/*                           GODOT ENGINE                                */
/*                      https://godotengine.org                          */
/*************************************************************************/
/* Copyright (c) 2007-2018 Juan Linietsky, Ariel Manzur.                 */
/* Copyright (c) 2014-2018 Godot Engine contributors (cf. AUTHORS.md)    */
/*                                                                       */
/* Permission is hereby granted, free of charge, to any person obtaining */
/* a copy of this software and associated documentation files (the       */
/* "Software"), to deal in the Software without restriction, including   */
/* without limitation the rights to use, copy, modify, merge, publish,   */
/* distribute, sublicense, and/or sell copies of the Software, and to    */
/* permit persons to whom the Software is furnished to do so, subject to */
/* the following conditions:                                             */
/*                                                                       */
/* The above copyright notice and this permission notice shall be        */
/* included in all copies or substantial portions of the Software.       */
/*                                                                       */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,       */
/* EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF    */
/* MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.*/
/* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY  */
/* CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,  */
/* TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE     */
/* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.                */
/*************************************************************************/

/**
	@author AndreaCatania
*/

/**
 * This is writtein in Cuda C
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "../../../thirdparty/flex/include/NvFlex.h"
#include "../../../thirdparty/flex/core/maths.h"

static const int kNumThreadsPerBlock = 256;

struct GdFlexExtCollisionVerifierCallback{
	NvFlexSolver* solver;

	GdFlexExtCollisionVerifierCallback(NvFlexSolver* p_solver)
		: solver(p_solver)
	{}
};

GdFlexExtCollisionVerifierCallback* GdFlexExtCreateCollisionVerifierCallback(NvFlexSolver* p_solver){
	return new GdFlexExtCollisionVerifierCallback(p_solver);
}

void GdFlexExtDestroyCollisionVerifierCallback(GdFlexExtCollisionVerifierCallback* p_callback){
	return delete p_callback;
}

__global__ void CollisionVerify(int numParticles, const Vec4* __restrict__ positions, Vec4* __restrict__ velocities, float dt){

	const int i = blockIdx.x*blockDim.x + threadIdx.x;
	const int particle_index = i;



}

void CollisionVerifierCallback(NvFlexSolverCallbackParams p_params){
	const int particle_count = p_params.numActive;

	const int kNumBlocks = (particle_count+kNumThreadsPerBlock-1)/kNumThreadsPerBlock;

	CollisionVerify<<<kNumBlocks, kNumThreadsPerBlock>>>(p_params.numActive, (Vec4*)p_params.particles, (Vec4*)p_params.velocities, p_params.dt);
}

void GdFlexExtSetCollisionVerifierCallback(GdFlexExtCollisionVerifierCallback *p_callback){
	NvFlexSolverCallback callback;
	callback.function = CollisionVerifierCallback;
	NvFlexRegisterSolverCallback(p_solver, callback, eNvFlexStageUpdateEnd);
}

__global__ void ComputeAABB(){

}

void ComputeAABBCallback(NvFlexSolverCallbackParams p_params){
	const int particle_count = p_params.numActive;

	const int kNumBlocks = (particle_count+kNumThreadsPerBlock-1)/kNumThreadsPerBlock;

	ComputeAABB<<<kNumBlocks, kNumThreadsPerBlock>>>();

}

void GdFlexExtSetComputeAABBCallback(NvFlexSolver* p_solver){
	NvFlexSolverCallback callback;
	callback.function = CollisionVerifierCallback;
	NvFlexRegisterSolverCallback(p_solver, callback, eNvFlexStageUpdateEnd);
}
