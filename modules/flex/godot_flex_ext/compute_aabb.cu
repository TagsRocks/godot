#include "hip/hip_runtime.h"
/*************************************************************************/
/*  collision_verifier.cu                                                */
/*************************************************************************/
/*                       This file is part of:                           */
/*                           GODOT ENGINE                                */
/*                      https://godotengine.org                          */
/*************************************************************************/
/* Copyright (c) 2007-2018 Juan Linietsky, Ariel Manzur.                 */
/* Copyright (c) 2014-2018 Godot Engine contributors (cf. AUTHORS.md)    */
/*                                                                       */
/* Permission is hereby granted, free of charge, to any person obtaining */
/* a copy of this software and associated documentation files (the       */
/* "Software"), to deal in the Software without restriction, including   */
/* without limitation the rights to use, copy, modify, merge, publish,   */
/* distribute, sublicense, and/or sell copies of the Software, and to    */
/* permit persons to whom the Software is furnished to do so, subject to */
/* the following conditions:                                             */
/*                                                                       */
/* The above copyright notice and this permission notice shall be        */
/* included in all copies or substantial portions of the Software.       */
/*                                                                       */
/* THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,       */
/* EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF    */
/* MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.*/
/* IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY  */
/* CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,  */
/* TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE     */
/* SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.                */
/*************************************************************************/

/**
	@author AndreaCatania
*/

/**
 * This is writtein in Cuda C
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "thirdparty/flex/include/NvFlex.h"
#include "thirdparty/flex/core/maths.h"

static const int kNumThreadsPerBlock = 256;

struct GdFlexExtComputeAABBCallback{
	NvFlexSolver *solver;

	int particle_body_count;
	float* aabbs;

	// Device memory
	int* d_last_pindex_particle_body; // Size of particle_body_count
	float* d_aabbs; // Size of particle_body_count * size(Vec3)


	GdFlexExtComputeAABBCallback(NvFlexSolver* p_solver)
		: solver(p_solver),
		  particle_body_count(0),
		  aabbs(NULL),
		  d_last_pindex_particle_body(NULL),
		  d_aabbs(NULL)
	{}

	~GdFlexExtComputeAABBCallback(){

		if(d_last_pindex_particle_body)
			hipFree(d_last_pindex_particle_body);

		if(d_aabbs)
			hipFree(d_aabbs);
	}
};

GdFlexExtComputeAABBCallback *GdFlexExtCreateComputeAABBCallback(NvFlexSolver *p_solver){
	return new GdFlexExtComputeAABBCallback(p_solver);
}

void GdFlexExtDestroyComputeAABBCallback(GdFlexExtComputeAABBCallback* p_callback){
	delete p_callback;
}

__global__ void ComputeAABB(const Vec4* __restrict__ positions, const int* __restrict__ p_sorted_to_original_map, int p_particle_body_count, int* __restrict__ p_last_pindex_particle_body, Vector3* p_aabbs){

	const int i = blockIdx.x * blockDim.x + threadIdx.x;
	const int sorted_particle_index = i;
	const int particle_index = p_sorted_to_original_map[sorted_particle_index];

	// Search the particle body owner
	int particle_body_index = -1;
	for( int y = 0; y < p_particle_body_count; ++y ){
		if( particle_index >= p_last_pindex_particle_body[y*2+0] && particle_index <= p_last_pindex_particle_body[y*2+1] ){
			particle_body_index = y;
			break;
		}
	}

	if( particle_body_index == -1 )
		return;

	const Vector3 other_vector = Vector3(positions[sorted_particle_index].x, positions[sorted_particle_index].y, positions[sorted_particle_index].z);

	// Perform AABB expand to other_vector
	// Godot AABB algorithm
	Vector3 begin = ((Vector3*)p_aabbs)[particle_body_index * 2 + 0];
	Vector3 end = begin + ((Vector3*)p_aabbs)[particle_body_index * 2 + 1];

	if (other_vector.x < begin.x)
		begin.x = other_vector.x;
	if (other_vector.y < begin.y)
		begin.y = other_vector.y;
	if (other_vector.z < begin.z)
		begin.z = other_vector.z;

	if (other_vector.x > end.x)
		end.x = other_vector.x;
	if (other_vector.y > end.y)
		end.y = other_vector.y;
	if (other_vector.z > end.z)
		end.z = other_vector.z;

	((Vector3*)p_aabbs)[particle_body_index*2+0] = begin;
	((Vector3*)p_aabbs)[particle_body_index*2+1] = end - begin;
}

void ComputeAABBCallback(NvFlexSolverCallbackParams p_params){

	GdFlexExtComputeAABBCallback* callback = static_cast<GdFlexExtComputeAABBCallback*>(p_params.userData);

	const int particle_count = p_params.numActive;

	// Using only one thread to avoid race condition
	// It's really limiting
	//const int kNumBlocks = (particle_count + kNumThreadsPerBlock - 1) / kNumThreadsPerBlock;
	//ComputeAABB<<<kNumBlocks, kNumThreadsPerBlock>>>(
	ComputeAABB<<<particle_count, 1>>>(
		(Vec4*)p_params.particles,
		p_params.sortedToOriginalMap,
		callback->particle_body_count,
		callback->d_last_pindex_particle_body,
		(Vector3*)callback->d_aabbs);

	hipMemcpy(callback->aabbs, callback->d_aabbs, sizeof(float) * 2 * 3 * callback->particle_body_count, hipMemcpyDeviceToHost);

	callback->aabbs = NULL;
}

void GdFlexExtSetComputeAABBCallback(GdFlexExtComputeAABBCallback* p_callback, int p_particle_body_count, int* p_last_pindex_particle_body, float* p_aabbs){

	p_callback->aabbs = p_aabbs;

	if( p_callback->particle_body_count != p_particle_body_count ){

		p_callback->particle_body_count = p_particle_body_count;

		if( p_callback->d_last_pindex_particle_body )
			hipFree(p_callback->d_last_pindex_particle_body);

		if( p_callback->d_aabbs )
			hipFree(p_callback->d_aabbs);

		hipMalloc(&p_callback->d_last_pindex_particle_body, sizeof(int) * p_particle_body_count * 2);
		hipMalloc(&p_callback->d_aabbs, sizeof(float) * 2 * 3 * p_particle_body_count);
	}

	hipMemcpyAsync(p_callback->d_last_pindex_particle_body, p_last_pindex_particle_body, sizeof(int) * p_particle_body_count * 2, hipMemcpyHostToDevice);
	hipMemcpyAsync(p_callback->d_aabbs, p_aabbs, sizeof(float) * 2 * 3 * p_particle_body_count, hipMemcpyHostToDevice);

	NvFlexSolverCallback solver_callback;
	solver_callback.function = ComputeAABBCallback;
	solver_callback.userData = p_callback;

	NvFlexRegisterSolverCallback(p_callback->solver, solver_callback, eNvFlexStageUpdateEnd);
}
